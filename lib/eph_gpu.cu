
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>

__global__
void dummy_test(int myID, int nrPS)
{
  int thread_index = threadIdx.x;
  int block_dimension = blockDim.x;
  int grid_dimension = gridDim.x;

  printf("%d %d %d %d %d\n", myID, nrPS, thread_index, block_dimension, grid_dimension);
}


void run_dummy_test_cu(int myID, int nrPS) 
{
  dummy_test<<<1, 1>>>(myID, nrPS);
}

